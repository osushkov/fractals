#include "hip/hip_runtime.h"

#include "MandelbrotKernel.hpp"
#include "CuVector.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <iostream>

static int *deviceItersOut = NULL;
static int *hostItersOut = NULL;

static size_t itersDataSize = 0;

__global__ void mandelbrotIters(
    int samplesX, int samplesY,
    double pWidth, double pHeight,
    CuVector2 minPixel, int iterations,
    int *itersOut) {

  int index = blockDim.x * blockIdx.x + threadIdx.x;

  int x = index % samplesX;
  int y = index / samplesX;

  if (y < samplesY) {
    CuVector2 p;
    p.x = minPixel.x + x * pWidth;
    p.y = minPixel.y + y * pHeight;

    int result = 0;

    CuVector2 cv;
    cv.x = 0.0;
    cv.y = 0.0;

    for (int i = 1; i <= iterations; i++) {
      double nx = cv.x*cv.x - cv.y*cv.y + p.x;
      double ny = 2.0*cv.x*cv.y + p.y;

      cv.x = nx;
      cv.y = ny;

      double length2 = nx*nx + ny*ny;
      result = (result == 0 && length2 >= 4.0) ? i : result;
    }

    itersOut[index] = result;
  }
}


std::vector<int> MandelbrotKernel::calculateIterationsPerPixel(
    int samplesX, int samplesY,
    double pWidth, double pHeight, CuVector2 minPixel,
    int iters) {

  int numPoints = samplesX * samplesY;

  size_t newItersDataSize = numPoints * sizeof(int);

  if (newItersDataSize > itersDataSize) {
    if (hostItersOut != NULL) {
      hipFree(deviceItersOut);
      free(hostItersOut);
    }

    itersDataSize = newItersDataSize;

    hostItersOut = (int *) malloc(itersDataSize);
    hipMalloc((void **)&deviceItersOut, itersDataSize);
  }

  int threadsPerBlock = 64;
  int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;

  mandelbrotIters<<<blocksPerGrid, threadsPerBlock>>>(
    samplesX, samplesY, pWidth, pHeight, minPixel, iters, deviceItersOut);

  hipMemcpy(hostItersOut, deviceItersOut, itersDataSize, hipMemcpyDeviceToHost);

  std::vector<int> result;
  for (int i = 0; i < numPoints; i++) {
    result.push_back(hostItersOut[i]);
  }

  return result;
}
